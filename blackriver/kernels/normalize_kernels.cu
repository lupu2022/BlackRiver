#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cstddef>


#include "common.h"
#include "block_reduce.h"
#include "kernels.h"

#define TILE_DIM 32

namespace cg = cooperative_groups;
namespace kernels {

template <typename T>
__forceinline__ __device__ T add_eps(T x, float eps) {
  return fabsf(x) > eps ? x : (x < 0 ? -eps : eps);
}


/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size* seq_len, hidden_size], ln result.
vars: [batch_size* seq_len], variance per token
means: [batch_size* seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_size], ln input.
scale: [hidden_size], ln scale
bias: [hidden_size], ln bias
*/
__global__ void ker_layer_norm(float *ln_res, float *vars, float *means, const float *inp,
                               const float *scale, const float *bias, int hidden_size, float eps) {
  // step 0. compute local sum
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 =
      reinterpret_cast<const float4 *>(inp) + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_square_sum +=
        val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
  }

  // step 1. compute reduce sum
  float mean_dim = float(hidden_size) * 4.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + eps;
    if (vars != nullptr) {
        vars[blockIdx.x] = s_var;
    }
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // step 2. layer norm result
  float4 *output_f4 =
      reinterpret_cast<float4 *>(ln_res) + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 vscale = __ldg(reinterpret_cast<const float4 *>(scale) + idx);
    float4 vbias = __ldg(reinterpret_cast<const float4 *>(bias) + idx);
    float4 val = inp_f4[idx];
    val.x = (val.x - s_mean) * s_var * vscale.x + vbias.x;
    val.y = (val.y - s_mean) * s_var * vscale.y + vbias.y;
    val.z = (val.z - s_mean) * s_var * vscale.z + vbias.z;
    val.w = (val.w - s_mean) * s_var * vscale.w + vbias.w;
    output_f4[idx] = val;
  }
}

void launch_layer_norm_float(float *ln_res, float *vars, float *means,
                          const float *inp, const float *scale,
                          const float *bias, int batch_size, int hidden_dim, float eps, 
                          hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
      throw ::std::runtime_error("violate hidden_dim % 4(float) 8(__half) = 0");
  }
  hidden_dim = hidden_dim >> 2;

  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
      ln_res, vars, means, inp, scale, bias, hidden_dim, eps);
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad,
                               const T *residual_grad, const T *inp_or_out,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means, const uint8_t *cmask,
                               int hidden_dim, float eps) {
  int offset = blockIdx.x * hidden_dim + threadIdx.x;
  float4 dxhat, xhat;
  float var_rsqrt;
  float temp_cmax_g;
  if (threadIdx.x < hidden_dim) {
    // step 0. dxhat = dout * gamma
    dxhat = ((const float4 *)out_grad)[offset];
    if (cmask) {
      uint32_t cmask4 = reinterpret_cast<const uint32_t *>(cmask)[offset];
      uint8_t *cm = reinterpret_cast<uint8_t *>(&cmask4);
      clip_bwd(dxhat.x, temp_cmax_g, dxhat.x, cm[0], 2);
      clip_bwd(dxhat.y, temp_cmax_g, dxhat.y, cm[1], 2);
      clip_bwd(dxhat.z, temp_cmax_g, dxhat.z, cm[2], 2);
      clip_bwd(dxhat.w, temp_cmax_g, dxhat.w, cm[3], 2);
    }
    float4 vgamma = ((const float4 *)gamma)[threadIdx.x];
    dxhat.x *= vgamma.x;
    dxhat.y *= vgamma.y;
    dxhat.z *= vgamma.z;
    dxhat.w *= vgamma.w;

    /*
    step 1. xhat = (output - betta) / gamma or
    (input - mean) * rsqrtf(var)
    */
    xhat = ((const float4 *)inp_or_out)[offset];
    var_rsqrt = rsqrtf((float)vars[blockIdx.x] + eps);
    if (means == nullptr) {
      // inp_or_out is output, xhat = (output - betta) / gamma
      float4 vbetta = ((const float4 *)betta)[threadIdx.x];
      xhat.x = (xhat.x - vbetta.x) / add_eps(vgamma.x, eps);
      xhat.y = (xhat.y - vbetta.y) / add_eps(vgamma.y, eps);
      xhat.z = (xhat.z - vbetta.z) / add_eps(vgamma.z, eps);
      xhat.w = (xhat.w - vbetta.w) / add_eps(vgamma.w, eps);
    } else {
      // inp_or_out is input, xhat = (input - mean) * rsqrtf(var)
      float fmean = (float)means[blockIdx.x];
      xhat.x = (xhat.x - fmean) * var_rsqrt;
      xhat.y = (xhat.y - fmean) * var_rsqrt;
      xhat.z = (xhat.z - fmean) * var_rsqrt;
      xhat.w = (xhat.w - fmean) * var_rsqrt;
    }
  }

  /* step2. block reduce sum for dxhat and dxhat*xhat */
  float reduce_val[2] = {0.f, 0.f};
  if (threadIdx.x < hidden_dim) {
    reduce_val[0] = dxhat.x + dxhat.y + dxhat.z + dxhat.w;
    reduce_val[1] = dxhat.x * xhat.x + dxhat.y * xhat.y + dxhat.z * xhat.z +
                    dxhat.w * xhat.w;
  }
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    float mean_dim = hidden_dim * 4;
    s_sum_dxhat = reduce_val[0] / mean_dim;
    s_sum_dxhat_xhat = reduce_val[1] / mean_dim;
  }
  __syncthreads();

  /*
  step3. compute input gradient
  (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / mean_dim) * rsqrt(var)
  */
  if (threadIdx.x >= hidden_dim) {
    return;
  }
  dxhat.x = (dxhat.x - s_sum_dxhat - xhat.x * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.y = (dxhat.y - s_sum_dxhat - xhat.y * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.z = (dxhat.z - s_sum_dxhat - xhat.z * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.w = (dxhat.w - s_sum_dxhat - xhat.w * s_sum_dxhat_xhat) * var_rsqrt;
  if (residual_grad) {
    // Add the residual grad,
    // usually in pre-layer-norm for transformer layer
    float4 dresidual = ((const float4 *)residual_grad)[offset];
    dxhat.x += dresidual.x;
    dxhat.y += dresidual.y;
    dxhat.z += dresidual.z;
    dxhat.w += dresidual.w;
  }
  ((float4 *)inp_grad)[offset] = dxhat;
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma


@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        T *cmax_grad, const T *out_grad,
                                        const T *inp_or_out, const T *gamma,
                                        const T *betta, const T *vars,
                                        const T *means, const uint8_t *cmask,
                                        int rows, int width, float eps) {
  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = threadIdx.y * width + idx;
  int y_stride = width * TILE_DIM;

  // Loop across inp height
  float dbetta = 0;
  float dgamma = 0;
  float dout, val;

  float thread_cmax_g = 0, cmax_g;
  if (idx < width) {
    if (means == nullptr) {
      float vbetta = (float)betta[idx];
      float vgamma = (float)gamma[idx];
      for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        dout = (float)out_grad[offset];
        if (cmask) {
          clip_bwd(dout, cmax_g, dout, cmask[offset], 2);
          thread_cmax_g += cmax_g;
        }
        // inp_or_out is output
        val = (float)inp_or_out[offset];
        dbetta += dout;
        dgamma += ((val - vbetta) / add_eps(vgamma, eps) * dout);
        offset += y_stride;
      }
    } else {
      for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        dout = (float)out_grad[offset];
        if (cmask) {
          clip_bwd(dout, cmax_g, dout, cmask[offset], 2);
          thread_cmax_g += cmax_g;
        }
        // inp_or_out is input
        val = (float)inp_or_out[offset];
        dbetta += dout;
        dgamma += ((val - (float)means[r]) *
                   rsqrtf((float)vars[r] + eps) * dout);
        offset += y_stride;
      }
    }
  }
  __shared__ float block_cmax_g;
  if (threadIdx.x == 0 && threadIdx.y == 0) block_cmax_g = 0;

  // Sum the shared buffer.
  betta_buffer[threadIdx.x][threadIdx.y] = dbetta;
  gamma_buffer[threadIdx.x][threadIdx.y] = dgamma;
  __syncthreads();

  if (thread_cmax_g != 0) {
    atomicAdd(&block_cmax_g, thread_cmax_g);
  }

  float s1 = betta_buffer[threadIdx.y][threadIdx.x];
  float s2 = gamma_buffer[threadIdx.y][threadIdx.x];
  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0) {
    if (cmask && block_cmax_g != 0) {
      atomicAdd(&cmax_grad[0], block_cmax_g);
    }
  }

  for (int i = 1; i < TILE_DIM; i <<= 1) {
    s1 += g.shfl_down(s1, i);
    s2 += g.shfl_down(s2, i);
  }

  int pos = blockIdx.x * TILE_DIM + threadIdx.y;
  if (threadIdx.x == 0 && idx < width) {
    betta_grad[pos] = s1;
    gamma_grad[pos] = s2;
  }
}



/**
Layer norm backword,
  compute the gradient of gamma, betta and input.
dbetta = sum(dout, dim=0)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
  (output - betta) / gamma if mean is nullptr
dgamma = sum(xhat * dout, dim=0)
dxhat = dout * gamma
dinp = (dxhat - (sum(dxhat, 1) + xhat * sum(dxhat * xhat, 1)) / hidden_dim)
  * rsqrt(var)

residual_grad, means, betta can be nullptr.
residual_grad will be added to dinp if it is not nullptr
  which is useful in transformer layer when pre-ln
means and betta are only used to compute xhat,
  (means == nullptr) ^ (betta == nullptr) should be true
*/
void launch_ln_bw_float(float *gamma_grad, float *betta_grad, float *inp_grad,
                  const float *out_grad, const float *residual_grad,
                  const float *inp_or_out, const float *gamma,
                  const float *betta, const float *vars,
                  const float *means, int batch, int hidden_dim, float eps,
                  hipStream_t stream[2]) {
  // compute grad of gamma and betta
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM) * TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream[0]>>>(
      gamma_grad, betta_grad, nullptr, out_grad, inp_or_out, gamma, betta, vars,
      means, nullptr, batch, hidden_dim, eps);
#if 0
  // compute grad of input
  if (hidden_dim % 4 != 0 || hidden_dim > 4096) {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 4096");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  ker_ln_bw_dinp<<<batch, nthread, 0, stream[1]>>>(
      inp_grad, out_grad, residual_grad, inp_or_out, gamma, betta, vars, means,
      nullptr, hidden_dim, eps);
#endif
}


}
